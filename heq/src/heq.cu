
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define TIMER_CREATE(t)               \
  hipEvent_t t##_start, t##_end;     \
  hipEventCreate(&t##_start);        \
  hipEventCreate(&t##_end);               
 
 
#define TIMER_START(t)                \
  hipEventRecord(t##_start);         \
  hipEventSynchronize(t##_start);    \
 
 
#define TIMER_END(t)                             \
  hipEventRecord(t##_end);                      \
  hipEventSynchronize(t##_end);                 \
  hipEventElapsedTime(&t, t##_start, t##_end);  \
  hipEventDestroy(t##_start);                   \
  hipEventDestroy(t##_end);     
  
#define TILE_SIZE 16
#define BLOCK_SIZE_X 1024 
#define BLOCK_SIZE_Y 1
#define BLOCK_SIZE_X2 1024
#define BLOCK_SIZE_Y2 1
#define CUDA_TIMING

unsigned char *input_gpu;
unsigned char *output_gpu;
unsigned int *hist;
unsigned char *lut;

double CLOCK() {
	struct timespec t;
	clock_gettime(CLOCK_MONOTONIC,  &t);
	return (t.tv_sec * 1000)+(t.tv_nsec*1e-6);
}

/*******************************************************/
/*                 Cuda Error Function                 */
/*******************************************************/
inline hipError_t checkCuda(hipError_t result) {
	#if defined(DEBUG) || defined(_DEBUG)
		if (result != cudaSuccess) {
			fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
			exit(-1);
		}
	#endif
		return result;
}
                
// Add GPU kernel and functions
__global__ void kernel(unsigned char *input, 
						unsigned int imgSize,
                       unsigned char *output){
        
  	int x = blockIdx.x*blockDim.x+threadIdx.x;
	int y = blockIdx.y*blockDim.y+threadIdx.y;
                
    int location = 	y*blockDim.x*gridDim.x+x;
	
    if (location<imgSize) output[location] = x%255;

}

__global__ void genHist(unsigned long long *input, unsigned int *hist) {

	int x = blockIdx.x*blockDim.x + threadIdx.x;

	unsigned long in=input[x];
	
	unsigned char temp[8];
	temp[0] = ((in & 0xFF00000000000000) >> 56);
	temp[1] = ((in & 0x00FF000000000000) >> 48);
	temp[2] = ((in & 0x0000FF0000000000) >> 40);
	temp[3] = ((in & 0x000000FF00000000) >> 32);
	temp[4] = ((in & 0x00000000FF000000) >> 24);
	temp[5] = ((in & 0x0000000000FF0000) >> 16);
	temp[6] = ((in & 0x000000000000FF00) >> 8);
	temp[7] = (in & 0x00000000000000FF);

	int count=1;
	unsigned char prev=temp[0];
	//Calculate Histogram
	for (int i=1; i<8; i++) {
		if (prev == temp[i]) count++;
		else {
			atomicAdd(&hist[prev], count);
			prev=temp[i];
			count=1;
		}
	}
	atomicAdd(&hist[prev], count);
	
}

__global__ void genHist2(unsigned char *input, int numPixel, unsigned int *hist){
	
  	int x = blockIdx.x*blockDim.x+threadIdx.x;
	      
	//Calculate Histogram
	if (x<numPixel){
		atomicAdd(&hist[input[x]], 1);
	}
}

__global__ void genLUT(unsigned int *hist, float imgSize, unsigned char *lut){
        
  	int x = blockIdx.x*blockDim.x+threadIdx.x;
	int y = blockIdx.y*blockDim.y+threadIdx.y;
                
    int location = 	y*blockDim.x*gridDim.x+x;

	__shared__ unsigned int cdfHist[256];
	__shared__ unsigned int tempHist[256];
	__shared__ int mincdf;

	tempHist[location]=hist[location];
	__syncthreads();

	//Accumulate
	unsigned int cdfTemp=0;	
	int i = location;
	do {
		cdfTemp += tempHist[i--];
	} while (i >= 0);
	cdfHist[location]=cdfTemp;
	__syncthreads();

	//Find minimum CDF
	if (threadIdx.x==0&&threadIdx.y==0) {
		int j=0;
		while (j<256 && cdfHist[j]==0) {
			++j;		
		}
		mincdf=j;
	}
	__syncthreads();

	//Generate look-up table
	float lutf=0;
	if (location>mincdf) {
		lutf=255.0*(cdfHist[location]-cdfHist[mincdf])/(imgSize-cdfHist[mincdf]);
	}
	//Write look-up table
	lut[location]=(unsigned char)roundf(lutf);
}

__global__ void applyLUT(unsigned int *input, unsigned int width, unsigned char *lut, unsigned int *output){
        
	int x = blockIdx.x*blockDim.x + threadIdx.x;

	__shared__ unsigned char lutTemp[256];
	lutTemp[threadIdx.x]=lut[threadIdx.x];
	__syncthreads();
	
	unsigned int temp=input[x];
	unsigned char temp1=lutTemp[(temp & 0xFF000000) >> 24];
	unsigned char temp2=lutTemp[(temp & 0x00FF0000) >> 16];
	unsigned char temp3=lutTemp[(temp & 0x0000FF00) >> 8];
	unsigned char temp4=lutTemp[(temp & 0x000000FF)];
	
	temp=(((unsigned int)temp1) << 24)+(((unsigned int)temp2) << 16)+(((unsigned int)temp3) << 8)+((unsigned int)temp4);

	output[x]=temp;
}

__global__ void applyLUT2(unsigned char *input, int numPixel, unsigned char *lut, unsigned char *output){
	
  	int x = blockIdx.x*blockDim.x+threadIdx.x;
	      
	//Generate new gray value
	if (x<numPixel){
		output[x]=lut[input[x]];
	}
}

void histogram_gpu(unsigned char *data, 
                   unsigned int height, 
                   unsigned int width){

	int gridXSize = width*height / BLOCK_SIZE_X;
	int gridYSize = 1;
                         
	int gridXSize2 = width*height / BLOCK_SIZE_X2;
	int gridYSize2 = 1;

	int restPixel = width*height % BLOCK_SIZE_X2;
	int lutOffset = gridXSize2 * BLOCK_SIZE_X2;
	
	// Both are the same size (CPU/GPU).
	unsigned int size = height*width;
	
	// Allocate arrays in GPU memory
	checkCuda(hipMalloc((void**)&input_gpu   , size*sizeof(unsigned char)));
	checkCuda(hipMalloc((void**)&output_gpu  , size*sizeof(unsigned char)));
	checkCuda(hipMalloc((void**)&hist  , 256*sizeof(unsigned int)));
	checkCuda(hipMalloc((void**)&lut  , 256*sizeof(unsigned char)));
	
    checkCuda(hipMemset(hist , 0 , 256*sizeof(unsigned int)));
    checkCuda(hipMemset(lut , 0 , 256*sizeof(unsigned char)));
	checkCuda(hipMemset(output_gpu , 0 , size*sizeof(unsigned char)));
				
    // Copy data to GPU
    checkCuda(hipMemcpy(input_gpu,	data, size*sizeof(char), hipMemcpyHostToDevice));

	checkCuda(hipDeviceSynchronize());
        
    // Execute algorithm
        
	dim3 dimGridforLUT(1, 1);
    dim3 dimBlockforLUT(16, 16);
	dim3 dimGrid2(gridXSize2, gridYSize2);
	dim3 dimBlock2(BLOCK_SIZE_X2/4, BLOCK_SIZE_Y2);

        // Kernel Call
	#if defined(CUDA_TIMING)
		float Ktime;
		TIMER_CREATE(Ktime);
		TIMER_START(Ktime);
	#endif
        
	dim3 dimGrid(gridXSize, gridYSize);
    dim3 dimBlock(BLOCK_SIZE_X/8, BLOCK_SIZE_Y);
    
    genHist<<<dimGrid, dimBlock>>>((unsigned long long*)input_gpu, hist);
    
    if (restPixel != 0 && size < 1024*8){
		int gridXSize3 = (restPixel-1) / (BLOCK_SIZE_X2/4) + 1;
		int gridYSize3 = 1;
		dim3 dimGrid3(gridXSize3, gridYSize3);

		genHist2<<<dimGrid3, dimBlock2>>>(input_gpu+lutOffset, restPixel, hist);
		genLUT<<<dimGridforLUT, dimBlockforLUT>>>(hist, size, lut);
	}
    
    else{
    	genLUT<<<dimGridforLUT, dimBlockforLUT>>>(hist, gridXSize*BLOCK_SIZE_X, lut);
	}

	applyLUT<<<dimGrid2, dimBlock2>>>((unsigned int*)input_gpu, width, lut, (unsigned int*)output_gpu);       

    if (restPixel != 0){
		int gridXSize3 = (restPixel-1) / (BLOCK_SIZE_X2/4) + 1;
		int gridYSize3 = 1;
		dim3 dimGrid3(gridXSize3, gridYSize3);

		applyLUT2<<<dimGrid3, dimBlock2>>>(input_gpu+lutOffset, restPixel, lut,	output_gpu+lutOffset);
	}
                                             
    checkCuda(hipDeviceSynchronize());
	
	#if defined(CUDA_TIMING)
		TIMER_END(Ktime);
		printf("Kernel Execution Time: %f ms\n", Ktime);
	#endif
        
	// Retrieve results from the GPU
	checkCuda(hipMemcpy(data, 
			output_gpu, 
			size*sizeof(unsigned char), 
			hipMemcpyDeviceToHost));
                        
        // Free resources and end the program
	checkCuda(hipFree(output_gpu));
	checkCuda(hipFree(input_gpu));
	checkCuda(hipFree(hist));
	checkCuda(hipFree(lut));

}

void histogram_gpu_warmup(unsigned char *data, 
                   unsigned int height, 
                   unsigned int width){
                         
	int gridXSize = 1 + (( width - 1) / TILE_SIZE);
	int gridYSize = 1 + ((height - 1) / TILE_SIZE);
	
	// Both are the same size (CPU/GPU).
	unsigned int size = height*width;
	
	// Allocate arrays in GPU memory
	checkCuda(hipMalloc((void**)&input_gpu   , size*sizeof(unsigned char)));
	checkCuda(hipMalloc((void**)&output_gpu  , size*sizeof(unsigned char)));
	
        checkCuda(hipMemset(output_gpu , 0 , size*sizeof(unsigned char)));
				
        // Copy data to GPU
        checkCuda(hipMemcpy(input_gpu, 
			data, 
			size*sizeof(char), 
			hipMemcpyHostToDevice));

	checkCuda(hipDeviceSynchronize());
        
        // Execute algorithm
        
		dim3 dimGrid(gridXSize, gridYSize);
        dim3 dimBlock(TILE_SIZE, TILE_SIZE);
        
        kernel<<<dimGrid, dimBlock>>>(input_gpu, 
										size,
                                      output_gpu);
                                             
        checkCuda(hipDeviceSynchronize());
        
	// Retrieve results from the GPU
	checkCuda(hipMemcpy(data, 
			output_gpu, 
			size*sizeof(unsigned char), 
			hipMemcpyDeviceToHost));
                        
        // Free resources and end the program
	checkCuda(hipFree(output_gpu));
	checkCuda(hipFree(input_gpu));

}

